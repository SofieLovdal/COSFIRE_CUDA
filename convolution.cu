
#include <hip/hip_runtime.h>
/*
* CUDA kernel for convolution in 2D, corresponding to conv2 in Matlab
* Sofie Lovdal 5.6.2018
*/

__global__ void conv2(double * output, double * const input, unsigned int const numRows, 
					  unsigned int const numCols, double * const kernel,
					  unsigned int const height_kernel, unsigned int const width_kernel)
{
   //width = numCols, height = numRows
   
   /*global thread ID in x dimension - moving horizontally in the image*/
   const int colIdx = blockIdx.x*blockDim.x + threadIdx.x;
   /*global thread ID in y dimension - moving vertically in the image*/
   const int rowIdx = blockIdx.y*blockDim.y + threadIdx.y;
   
   int i, j, kernelIdx, imageIdx;
    
   /*make sure we are within image*/
   if(colIdx>=numCols || rowIdx >= numRows) return; 
   
   /*Linear index of pixel corresponding to current thread */
   int linearIdx = rowIdx*numCols + colIdx;
   
   int kernel_radius=height_kernel/2;
   int imageRowIdx, imageColIdx;
   
   /*Apply convolution to linarIdx (pixel that each thread should treat) */
	double sum=0.0;
	for (i = -kernel_radius; i <= kernel_radius; i++) {
		for (j = -kernel_radius; j <= kernel_radius; j++) {
			kernelIdx = width_kernel*(i+kernel_radius) + (j+kernel_radius);
			imageRowIdx = rowIdx+i;
			imageColIdx = colIdx+j;
			imageIdx = imageRowIdx*numCols + imageColIdx;
			/*zero padding at borders: top, bottom, left, right*/
			if(imageRowIdx<0 || imageRowIdx >=numRows || imageColIdx <0 || imageColIdx >= numCols ) {
				sum+=0.0;
			} else {	
				sum=sum+input[imageIdx]*kernel[kernelIdx];
			}
		}	
	}
	output[linearIdx] = sum;
}
