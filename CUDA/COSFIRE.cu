#include "hip/hip_runtime.h"
/*
* CUDA kernel for effective implementation of the COSFIRE algorithm.
* 
* Takes a preprocessed input image and a set of tuples as input (preprocessing
* of image and configuration of filter is done in Matlab).
* The kernel performs the following steps in parallel:
* 
* 1. Generates DoG filters corresponding to each unique sigma
* and convolves this with the input image
* 
* 2. Generates Gaussian blur filter for each unique (sigma, rho)-combination
* in the set of tuples and performs dilation (max-blurring) on each corresponding
* response from (1.)
* 
* 3.Output the response image by weighted geometric mean after shifting 
* each subresponse according to (rho, theta).
* 
* Sofie Lovdal 5.6.2018
* 
* For now, the implementation allocates the necessary big chunks of 
* memory on host side and passes it to the algorithm governing kernel.
*/
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "getDoG.cu"
#include "convolution.cu"
#include "maxBlur.cu"
#include "shiftPixels.cu"
#include "geometricMean.cu"
#include "getGaussian.cu"

/*some maximum size for some buffers*/
__constant__ int MAXSIZE=900;

__global__ void COSFIRE_CUDA(double * output, double * const input,
					unsigned int const numRows, unsigned int const numCols,
					double * tuples, unsigned int const numTuples,
					double * responseBuffer1, double * responseBuffer2,
					double const threshold, double const sigmaratio,
					double const alpha, double const sigma0)
{	   
   /*Maximize GPU load. Sync before output merging*/
   /*An idea would be transposing the input matrix to begin with, then we get rid of the column first order problem*/
   
   /*The dynamic parallelism of the kernel is structured as follwing: 
    * One thread for each tuple is launched from host side.
    * Thread i launches workflow for single tuple (outputresponse)
    */
    
    /*As many threads for this kernels are launched as the number of tuples: initial thread pool is 1D array so thread ID is threadIdx.x*/
    /*we create a pointer to each thread's place in the array so that we can pass this as argument to functions*/
    hipError_t err;
    
    double * myTuple = &(tuples[3*threadIdx.x]);
    double * myResponse1 = &(responseBuffer1[numRows*numCols*threadIdx.x]);
    double * myResponse2 = &(responseBuffer2[numRows*numCols*threadIdx.x]);
    
    double * DoGfilter;
    DoGfilter = (double*)malloc(MAXSIZE*sizeof(double));
	double mySigma = myTuple[0];

	int sz = ceil(mySigma*3) * 2 + 1; //related to calculating suitable block size for getDoG kernel launch
	dim3 gridSize (1);
	dim3 blockSize (sz, sz, 1);
    getDoG<<<1, blockSize>>>(DoGfilter, mySigma, sigmaratio); //launch one grid with blocksize sz. Every tuple-thread does this - dynamic parallelism.
	err = hipGetLastError();
    if ( hipSuccess != err )
    {
       //printf("cudaCheckError() failed at COSFIRE_CUDA call %s\n", hipGetErrorString( err ) );
    }
    __syncthreads();
	hipDeviceSynchronize();

	dim3 blockSize2 (16, 16, 1);
    dim3 gridSize2 (ceil((double)numRows/16), ceil((double)numCols/16));
	
	conv2<<<gridSize2, blockSize2>>>(myResponse1, input, numRows, numCols, DoGfilter, sz, sz);
    
	err = hipGetLastError();
    if ( hipSuccess != err )
    {
       //printf("cudaCheckError() failed at COSFIRE_CUDA call %s\n", hipGetErrorString( err ) );
    }
    
    __syncthreads();
	hipDeviceSynchronize();
	
	double rho = myTuple[1];
	double blurSigma = sigma0 + alpha*rho; //CHANGE SIZE OF FILTER + NO NORMALIZATION OF VALUES
	sz = ceil(blurSigma*3.0)*2+1;
	dim3 blockSize3(sz, sz, 1);
	/*Here: control proper size of 2D Gaussian filter. Matlab code does this with separable filters I believe so
	 * I cannot directly compare. So here send filter with blurSigma to maxBlur??*/
	 
	 /*Something goes wrong here: Thread 0 gives empty output (black, all zeros), while it does produce OKAY results for the
	  * other threads. Sz is apparently 1, and blurSigma 0.*/

	   getGaussian<<<1, blockSize3>>>(DoGfilter, blurSigma);
	   //output[25]=(double)sz;
	   //output[26]=blurSigma;
	   //output[27]=sigma0;
	   //output[28]=alpha;
	   //output[29]=rho;
	   //output[30]=5;
       maxBlur<<<gridSize2, blockSize2>>>(myResponse2, myResponse1, numRows, numCols, DoGfilter, sz, sz);
	//launch Kernel that inserts the DoG convoluted with input into myResponse (write this control flow kernel) + sync
	//launch Kernel that inserts the Gaussian maxblurring into another buffer (myResponse_maxBlur)? + sync
	//launch Kernel that shifts pixels from maxBlur buffer into new buffer (we can reuse myResponse now I guess)
	//master thread can launch kernel for geometricMean of myResponse, put into output.
	
	   
   double phi = myTuple[2];
   //something fishy going on with the shifting. fix this
   if(threadIdx.x==5) {
   shiftPixels<<<gridSize2, blockSize2>>>(output, myResponse2, numRows, numCols, rho, phi);
	}
    __syncthreads();
	hipDeviceSynchronize();
   
   if(threadIdx.x == 0) {
	   //geometricMean<<<gridSize2, blockSize2>>>(output, responseBuffer1, numRows, numCols, numTuples, threshold);
   }	    
   
   /*Launch getDoG kernel for each sigma in set S!
    * The ideal amount of threads for this kernel is sz*sz, 
    *Return the 2D DoG filter which is then convolved here with input image*/
   
   /*Convolve with input -- separable filters?? */
   
   /*Create Gaussian blur filter for each rho-sigma combination*/
   
   /*Convolve (max-blurring) with each corresponding response from DoG convolution*/
   
   /*Obtain final response by inspecting subresponses (array of 2D matrices
    * and their corresponding shift info is needed)*/

}

