
#include <hip/hip_runtime.h>
/*
* CUDA kernel for geometric mean for calculating response of COSFIRE filter
* Sofie Lovdal 18.6.2018
* The input is a flattened 3D array of all responses obtained from the COSFIRE
* algorithm. The argument output is a buffer for the final response, input is a 1D
* array of dimensions numResponses*rumRows*numCols.
*/

__global__ void geometricMean(double * output, double * const input, 
					  unsigned int const numRows, unsigned int const numCols, 
					  int const numResponses, double const threshold)
{
   
   const int colIdx = blockIdx.x*blockDim.x + threadIdx.x;
   const int rowIdx = blockIdx.y*blockDim.y + threadIdx.y;
    
   /*make sure we are within image*/
   if(colIdx>=numCols || rowIdx >= numRows) return; 
   
   /*Pixel to consider in outputimage*/
   int linearIdx = rowIdx*numCols + colIdx;
   
   double product=1.0;
   int i;
   for(i=0; i<numResponses; i++) {
	   product*=input[linearIdx+i*numRows*numCols];
   }
   
   double mu = pow(product, (1.0/(double)numResponses));
   output[linearIdx] = mu; //here: (no)thresholding
}
